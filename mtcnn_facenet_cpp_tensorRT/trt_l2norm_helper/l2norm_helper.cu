#include "hip/hip_runtime.h"
#include "l2norm_helper.h"


template<typename T>
__global__ void sqrtKernel(const int n, const T* x, T* y)
{
    printf("Unsupported type.");
}

template<>
__global__ void sqrtKernel(
    const int n,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = sqrtf(x[i]);
    }
}

template<>
__global__ void sqrtKernel(
    const int n,
    const __half* x,
    __half* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
         y[i] = hsqrt(x[i]);
    }
}

template<typename T>
__global__ void rsqrtKernel(const int n, const T* x, T* y)
{
    printf("Unsupported type.");
}

template<>
__global__ void rsqrtKernel(
    const int n,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = rsqrtf(x[i]);
    }
}

template<>
__global__ void rsqrtKernel(
    const int n,
    const __half* x,
    __half* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
         y[i] = hrsqrt(x[i]);
    }
}

template<typename T>
__global__ void maxKernel(const int n, const T eps, const T* x, T* y)
{
    printf("Unsupported type.");
}

template<>
__global__ void maxKernel(
    const int n,
    const float eps,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = fmaxf(x[i], eps);
    }
}

template<>
__global__ void maxKernel(
    const int n,
    const __half eps,
    const __half* x,
    __half* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        if (__hgt(x[i], eps))
        {
          y[i] = x[i];
        } else {
          y[i] = eps;
        }
    }
}

template <typename T>
bool executeInference(
    hipStream_t stream,
    int op_type,
    T eps,
    int batch_size,
    int C,
    int H,
    int W,
    const T* input,
    T* output)
{
    const int length = C * H * W;
    for (int n = 0; n < batch_size; ++n)
    {
        switch(op_type)
        {
          case operation_t::OP_TYPE_MAX:
            maxKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, eps, input, output);
            break;
          case operation_t::OP_TYPE_RSQRT:
            rsqrtKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, input, output);
            break;
          case operation_t::OP_TYPE_SQRT:
            sqrtKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, input, output);
            break;
          default:
            return 1;
        }
        // Move cursors
        input += length;
        output += length;
    }
    return 0;
}

int L2NormHelper::enqueue(
    int batchSize,
    const void* const* inputs,
    void** outputs,
    void* workspace,
    hipStream_t stream)
{
    switch(mDataType)
    {
      case DataType::kFLOAT:
        if (!executeInference(stream, op_type, eps, batchSize, C, H, W,
                              (const float*)inputs[0], (float*)outputs[0]))
          {
            return 1;
          }
        break;
      case DataType::kHALF:
        if (!executeInference(stream, op_type, (__half)eps, batchSize, C, H, W,
                              (const __half*)inputs[0], (__half*)outputs[0]))
          {
            return 1;
          }
        break;
      default:
        return 1;
    }
    return 0;
}
